#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel
__global__ void vector_add(const float *a, const float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    int n = 1 << 20;  // 1M elements
    size_t size = n * sizeof(float);

    // Allocate host memory
    float *h_a = new float[n];
    float *h_b = new float[n];
    float *h_c = new float[n];

    for (int i = 0; i < n; ++i) {
        h_a[i] = 1.0f;
        h_b[i] = 2.0f;
    }

    // Allocate device memory
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy data to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    vector_add<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Verify result
    for (int i = 0; i < 10; ++i) {
        std::cout << "c[" << i << "] = " << h_c[i] << std::endl;
    }

    // Cleanup
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}

